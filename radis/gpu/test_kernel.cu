#include "hip/hip_runtime.h"

#include "gpu_cpu_agnostic.h"

extern "C"{

struct myStruct {
    int n;
    float x;
    float y;
};


struct transform {
    int offset;
    int scale;
};


__device__ __constant__ transform params;
__device__ __constant__ myStruct struct_obj;
__device__ __constant__ size_t N;
__device__ __constant__ int Nt;
__device__ __constant__ int Nf;
__device__ __constant__ float dt;
__device__ __constant__ float wL;

__global__ void return_dims(int* a){
    a[0] = blockDim.x;
    a[1] = gridDim.x;

}

__global__ void add_ints(int* a, int* b, int* c){
    LOOP(threadIdx.x, blockDim.x){
        LOOP(blockIdx.x, gridDim.x){
            int i = threadIdx.x + blockDim.x * blockIdx.x;
            if (i < struct_obj.n){
                c[i] = a[i] + b[i];
            }
        }
    }
}


__global__ void matSum(int *a, int *b, int *c){
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = (a[tid] + b[tid]) * params.scale + params.offset;
    }


__global__ void applyLineshapes(complex<float>* data) {

    const float pi = 3.141592653589793f;

    LOOP(threadIdx.x, blockDim.x){
        LOOP(blockIdx.x, gridDim.x){
            int k = threadIdx.x + blockDim.x * blockIdx.x;
            if (k < Nf) {
                float x = k / (Nt * dt);
                float mul = 0.0;
                complex<float> out_complex = 0;

                mul = expf(- pi * x * wL);
                out_complex += mul * data[k];

                data[k].real(out_complex.real());
                data[k].imag(out_complex.imag());
            }
        }
    }
}


}
